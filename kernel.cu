#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "err_handler.h"
#include <stdio.h>


using namespace std;

__device__ int mytoupper(int a){
	if (a >= 'a' && a <= 'z' )
	return a-'a'+'A';

}
__global__ void kernel(char* input, char *result, int size)
{
	
    int i = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	
	while (i < size){
		//result[i] = mytoupper(input[i]);
		if (input[i] >= 'a' && input[i] <= 'z')
			result[i] = input[i] - 'a' + 'A';
		else result[i] = input[i];
		i += stride;
	}

}

int main()
{
	char input[] = "hello, cuda.";
	int size = sizeof(input);
	char *dev_input;
	char *dev_output;
	char *result;
	
	// copy string from host to device
	HANDLE_ERROR(hipMalloc((void**)&dev_input, size*sizeof(char)));
	HANDLE_ERROR(hipMemset(dev_input, 0, size*sizeof(char)));
	HANDLE_ERROR(hipMemcpy(dev_input, input, size*sizeof(char), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMalloc((void**)&dev_output, size*sizeof(char)));
	HANDLE_ERROR(hipMemset(dev_output, 0, size*sizeof(char)));

	result = (char*)malloc(size*sizeof(char));

	kernel<<<1, 256>>>(dev_input, dev_output,size);

	//copy result from device to host
	HANDLE_ERROR(hipMemcpy(result, dev_output, size*sizeof(char), hipMemcpyDeviceToHost));
	printf("result: %s \n", result);
	system("pause");
	// clean
	HANDLE_ERROR(hipFree(dev_input));
	HANDLE_ERROR(hipFree(dev_output));
	free(result);
	printf("done\n");
	system("pause");
    return 0;
}


